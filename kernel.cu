#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""


#define MAX 200
#define LEN 10000
#define SAM 100

#define TILE_WIDTH 32

using namespace std;

__global__ void Initialize(char* _A, int _nLength) {
	int i = blockIdx.x *blockDim.x + threadIdx.x;
	int j = blockIdx.y *blockDim.y + threadIdx.y;
	bool finding = true;
	int RndAsc = 0;
	while (finding) {
		RndAsc = (clock() + i + j) % ((123 - 65) + 1) + 65;
		if (RndAsc == 123) RndAsc = 32;
		if ((RndAsc <= 90) || (RndAsc >= 97)) finding = false;
	}
	_A[i*_nLength + j] = char(RndAsc);
}

void Evaluation(char* _A, char* _Input, int _nLength, int* _nSum) {
	for (int i = 0; i < MAX; i++) {
		for (int j = 0; j < _nLength; j++) {
			if (_A[i * _nLength + j] == _Input[j])
				_nSum[i]++;
		}
	}
}

int Selection(char* _A, char* _ABackUp, int _nLength, int* _nSum) {
	int *nSamList = new int[SAM];
	for (int i = 0; i < SAM; i++) {
		int max = 0;
		int temp = 0;
		for (int j = 0; j < MAX; j++) {
			if (max < _nSum[j]) {
				max = _nSum[j];
				temp = j;
			}
		}
		nSamList[i] = temp;
		_nSum[temp] = 0;
	}
	for (int i = 0; i < SAM; i++) {
		for (int j = 0; j < _nLength; j++) {
			_ABackUp[i*_nLength + j] = _A[nSamList[i] * _nLength + j];
		}
	}

	return nSamList[0];
}

__global__ void Replace(char* _A, char* _ABackUp, int _nLength) {
	int i = blockIdx.x *blockDim.x + threadIdx.x;
	if (i < MAX) {
		int nCut = (clock() + blockDim.x * threadIdx.x) % _nLength;
		int F = (clock() - blockDim.x + threadIdx.x) % SAM;
		int M = (clock() + blockDim.x - threadIdx.x) % SAM;
		for (int j = 0; j < nCut; j++) {
			_A[i * _nLength + j] = _ABackUp[F*_nLength + j];
		}
		for (int j = nCut; j < _nLength; j++) {
			_A[i * _nLength + j] = _ABackUp[M*_nLength + j];
		}
	}
}

__global__ void Mutation(char* _A, int _nLength) {
	int i = blockIdx.x *blockDim.x + threadIdx.x;
	int j = blockIdx.y *blockDim.y + threadIdx.y;
	if (i < MAX && j < _nLength) {
		if ((clock() + blockIdx.x - blockDim.y * threadIdx.x) % 1000 == 0) {
			bool finding = true;
			int RndAsc = 0;
			while (finding) {
				RndAsc = (clock() + blockIdx.y - blockDim.x * threadIdx.y) % ((123 - 65) + 1) + 65;
				if (RndAsc == 123) RndAsc = 32;
				if ((RndAsc <= 90) || (RndAsc >= 97)) finding = false;
			}
			_A[i * _nLength + j] = char(RndAsc);
		}
	}
}

void main() {
	char *A = new char[MAX*LEN];
	char *ABackUp = new char[MAX*LEN];

	char *dev_A, *dev_ABackUp;

	hipError_t cudaStatus = hipSetDevice(0);
	cudaStatus = hipMalloc((void**)&dev_A, MAX * LEN * sizeof(bool));
	cudaStatus = hipMalloc((void**)&dev_ABackUp, MAX * LEN * sizeof(bool));

	dim3 dimGrid((MAX - 1) / TILE_WIDTH + 1, (LEN - 1) / TILE_WIDTH + 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
	dim3 dimGrid2((MAX - 1) / TILE_WIDTH + 1);
	dim3 dimBlock2(TILE_WIDTH);

	srand(time(NULL));
	//memory allocation

	char Input[MAX] = { 0 };
	cout << "Input Sentece :";

	fgets(Input, sizeof(Input), stdin);
	//gene initialize

	int nLength = strlen(Input) - 1;
	clock_t st = clock();
	Initialize << <dimGrid, dimBlock >> > (dev_A, nLength);
	hipDeviceSynchronize();

	int stack = 0;
	cudaStatus = hipMemcpy(A, dev_A, MAX*LEN * sizeof(bool), hipMemcpyDeviceToHost);
	while (true) {
		int *nSum = new int[MAX] {0};
		//gene evaluation
		Evaluation(A, Input, nLength, nSum);
		int *nSumCopy = new int[MAX] {0};
		memcpy(nSumCopy, nSum, MAX * sizeof(int));
		//gene selection
		int nMaxGen = Selection(A, ABackUp, nLength, nSumCopy);
		stack++;

		if (stack % 1000 == 0) {
			cout << endl << stack << "��°\t" << endl;
			for (int i = 0; i < nLength; i++)
				cout << A[nMaxGen * nLength + i];
			cout << endl;
		}

		if (nSum[nMaxGen] >= nLength) {
			cout << endl << stack << "��°\t" << ((float)(clock() - st)) / 1000 << "��  \t" << endl << endl;
			cout << "���� ���" << endl;
			for (int i = 0; i < nLength; i++)
				cout << A[nMaxGen * nLength + i];
			cout << endl;
			break;
		}

		//gene replacement
		cudaStatus = hipMemcpy(dev_A, A, MAX*LEN * sizeof(bool), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_ABackUp, ABackUp, MAX*LEN * sizeof(bool), hipMemcpyHostToDevice);
		Replace << <dimGrid2, dimBlock2 >> >(dev_A, dev_ABackUp, nLength);
		hipDeviceSynchronize();

		//gene mutation
		Mutation << <dimGrid, dimBlock >> >(dev_A, nLength);
		hipDeviceSynchronize();

		cudaStatus = hipMemcpy(A, dev_A, MAX*LEN * sizeof(bool), hipMemcpyDeviceToHost);
		cudaStatus = hipMemcpy(ABackUp, dev_ABackUp, MAX*LEN * sizeof(bool), hipMemcpyDeviceToHost);
	}
}
